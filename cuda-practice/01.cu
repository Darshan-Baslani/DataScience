
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel(void) {
    printf("Hello, Cuda\n");
}

int main(void) {
    kernel<<< 1, 1 >>>();
    hipDeviceSynchronize();
    printf("Hello, world");
    return 0;
}